#include "hip/hip_runtime.h"
#include <util_log.h>
#include <util_solve_equation_choices.h>

#include <cusp/coo_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/precond/ainv.h>
#include <cusp/precond/diagonal.h>
#include <cusp/precond/aggregation/smoothed_aggregation.h>
#include <cusp/monitor.h>
#include <cusp/krylov/cg.h>
#include <cusp/krylov/cr.h>
#include <cusp/krylov/gmres.h>
#include <cusp/krylov/bicg.h>
#include <cusp/krylov/bicgstab.h>

#include <vector>

namespace util
{

  namespace detail
  {

    class DispatchSolver
    {
    protected:

      template<typename LinearOperator, typename Vector, typename Monitor, typename Preconditioner>
      void invoke_solver(
                         LinearOperator const & A
                         , Vector & x
                         , Vector & b
                         , Monitor & monitor
                         , Preconditioner & P
                         , unsigned int const & solver_choice
                         )
      {
        util::Log        logging;
        std::string const newline = util::Log::newline();

        switch (solver_choice)
        {
          case CG_SOLVER:
          {
            cusp::krylov::cg(A, x, b, monitor, P);
          }
            break;
          case CR_SOLVER:
          {
            cusp::krylov::cr(A, x, b, monitor, P);
          }
            break;
          case BICG_SOLVER:
          {
            cusp::krylov::bicg(A, A, x, b, monitor, P, P);
          }
            break;
          case BICGSTAB_SOLVER:
          {
            cusp::krylov::bicgstab(A, x, b, monitor, P);
          }
            break;
          case GMRES_SOLVER:
          {
            unsigned int restart = 10u;  // hardwired constant

            cusp::krylov::gmres(A, x, b, restart, monitor, P);
          }
            break;
          default:
            logging << "Unrecognized solver choice  " << solver_choice << newline;
            break;
        }
      }

    public:

      template<typename LinearOperator, typename Vector, typename Monitor >
      void solve(
                 LinearOperator const & A
                 , Vector & x
                 , Vector & b
                 , Monitor & monitor
                 , unsigned int const & precond_choice
                 , unsigned int const & solver_choice
                 )
      {
        typedef typename LinearOperator::value_type   T;
        typedef typename LinearOperator::index_type   I;
        typedef typename LinearOperator::memory_space M;

        util::Log        logging;
        std::string const newline = util::Log::newline();

        switch (precond_choice)
        {
          case DIAGONAL_PRECONDITIONER:
          {
            cusp::precond::diagonal<T, M> P(A);
            invoke_solver(A, x, b, monitor, P, solver_choice);
          }
            break;
          case IDENTITY_PRECONDITIONER:
          {
            cusp::identity_operator<T, M> P(A.num_rows, A.num_rows);
            invoke_solver(A, x, b, monitor, P, solver_choice);
          }
            break;
          case BRIDSON_AINV_PRECONDITIONER:
          {
            cusp::precond::bridson_ainv<T, M>  P(A, 0, -1, true, 2);
            invoke_solver(A, x, b, monitor, P, solver_choice);
          }
            break;
          default:
            logging << "Unrecognized preconditioner choice  " << precond_choice << newline;
            break;
        }
      }

    };

  }// namespace detail

  /**
   *
   * @tparam M2    The target memory space can be either cusp::host_memory or
   *               cusp::device_memory. This essesially controls whether a GPU
   *               or CPU computation is done.
   */
  template<typename M2,typename T>
  __host__ void solve_equation(
                                      unsigned int n
                                      , unsigned int k
                                      , std::vector<unsigned int> const & A_row_indices
                                      , std::vector<unsigned int> const & A_column_indices
                                      , std::vector<T           > const & A_values
                                      , std::vector<T           >       & x_values
                                      , std::vector<T           > const & b_values
                                      , unsigned int              const & max_iterations
                                      , T                         const & relative_tolerance
                                      , T                         const & absolute_tolerance
                                      , unsigned int              const & preconditioner_choice
                                      , unsigned int              const & solver_choice
                                      , std::vector<T>                  * residuals
                                      )
  {
    util::Log        logging;
    std::string const newline = util::Log::newline();

    //--- This simulator uses CUSP for number crunching. -------------------------
    //---
    //--- Read more about CUSP here
    //--- https://code.google.com/p/cusp-library/wiki/QuickStartGuide
    //---
    typedef unsigned int        I;
    typedef cusp::host_memory   M1;

    //--- Allocate storage for (n,n) matrix with k nonzeros ---------------------
    cusp::coo_matrix<I,  T,  M1>  A1(n,n,k);

    //--- Initialize matrix entries on host -------------------------------------
    A1.row_indices    = A_row_indices;
    A1.column_indices = A_column_indices;
    A1.values         = A_values;

    //--- Allocate space for two n-dimensional vectors and initialize them -------
    cusp::array1d<T, M1> x1(x_values);
    cusp::array1d<T, M1> b1(b_values);

    //--- Convert COO‐>CSR on the host and transfer to the device ----------------
    cusp::csr_matrix<I, T,  M2> A2  =  A1;

    //--- Transfer vectors to device ---------------------------------------------
    cusp::array1d<T, M2> x2 = x1;
    cusp::array1d<T, M2> b2 = b1;

    //--- Solve the linear system A * x = b with the Conjugate Gradient method ---
    cusp::convergence_monitor<T> monitor(b2, min( max_iterations  ,n), relative_tolerance, absolute_tolerance);

    detail::DispatchSolver dispatcher;

    dispatcher.solve(A2, x2, b2, monitor, preconditioner_choice, solver_choice);

    //--- Monitor will report solver progress and results ------------------------
    if (!monitor.converged())
    {
      logging << "Solver reached iteration limit " << monitor.iteration_limit() << " before converging";
      logging << " to " << monitor.tolerance() << " tolerance ";
      logging << " (" << monitor.residual_norm() << " final residual)" << newline;
    }

    //--- Read back solution from device to host ---------------------------------
    x1 = x2;
    thrust::copy( x1.begin(), x1.end(), x_values.begin() );

    if(residuals)
    {
      residuals->resize(monitor.residuals.size());
      thrust::copy( monitor.residuals.begin(), monitor.residuals.end(), residuals->begin() );
    }
  }

  __host__ void solve_equation_host(
                                           unsigned int n
                                           , unsigned int k
                                           , std::vector<unsigned int> const & A_row_indices
                                           , std::vector<unsigned int> const & A_column_indices
                                           , std::vector<float       > const & A_values
                                           , std::vector<float       >       & x_values
                                           , std::vector<float       > const & b_values
                                           , unsigned int              const & max_iterations
                                           , float                     const & relative_tolerance
                                           , float                     const & absolute_tolerance
                                           , unsigned int              const & preconditioner_choice
                                           , unsigned int              const & solver_choice
                                           , std::vector<float>              * residuals
                                           )
  {
    typedef cusp::host_memory   M2;

    solve_equation<M2,float>(
                             n
                             , k
                             , A_row_indices
                             , A_column_indices
                             , A_values
                             , x_values
                             , b_values
                             , max_iterations
                             , relative_tolerance
                             , absolute_tolerance
                             , preconditioner_choice
                             , solver_choice
                             ,residuals
                             );
  }

  __host__ void solve_equation_device(
                                             unsigned int n
                                             , unsigned int k
                                             , std::vector<unsigned int> const & A_row_indices
                                             , std::vector<unsigned int> const & A_column_indices
                                             , std::vector<float       > const & A_values
                                             , std::vector<float       >       & x_values
                                             , std::vector<float       > const & b_values
                                             , unsigned int              const & max_iterations
                                             , float                     const & relative_tolerance
                                             , float                     const & absolute_tolerance
                                             , unsigned int              const & preconditioner_choice
                                             , unsigned int              const & solver_choice
                                             , std::vector<float>              * residuals
                                             )
  {
    typedef cusp::device_memory M2;

    solve_equation<M2,float>(
                             n
                             , k
                             , A_row_indices
                             , A_column_indices
                             , A_values
                             , x_values
                             , b_values
                             , max_iterations
                             , relative_tolerance
                             , absolute_tolerance
                             , preconditioner_choice
                             , solver_choice
                             ,residuals
                             );
  }

  __host__ void solve_equation_host(
                                           unsigned int n
                                           , unsigned int k
                                           , std::vector<unsigned int> const & A_row_indices
                                           , std::vector<unsigned int> const & A_column_indices
                                           , std::vector<double      > const & A_values
                                           , std::vector<double      >       & x_values
                                           , std::vector<double      > const & b_values
                                           , unsigned int              const & max_iterations
                                           , double                    const & relative_tolerance
                                           , double                    const & absolute_tolerance
                                           , unsigned int              const & preconditioner_choice
                                           , unsigned int              const & solver_choice
                                           , std::vector<double>             * residuals
                                           )
  {
    typedef cusp::host_memory   M2;

    solve_equation<M2,double>(
                              n
                              , k
                              , A_row_indices
                              , A_column_indices
                              , A_values
                              , x_values
                              , b_values
                              , max_iterations
                              , relative_tolerance
                              , absolute_tolerance
                              , preconditioner_choice
                              , solver_choice
                              ,residuals
                              );
  }

  __host__ void solve_equation_device(
                                             unsigned int n
                                             , unsigned int k
                                             , std::vector<unsigned int> const & A_row_indices
                                             , std::vector<unsigned int> const & A_column_indices
                                             , std::vector<double      > const & A_values
                                             , std::vector<double      >       & x_values
                                             , std::vector<double      > const & b_values
                                             , unsigned int              const & max_iterations
                                             , double                    const & relative_tolerance
                                             , double                    const & absolute_tolerance
                                             , unsigned int              const & preconditioner_choice
                                             , unsigned int              const & solver_choice
                                             , std::vector<double>             * residuals
                                             )
  {
    typedef cusp::device_memory M2;

    solve_equation<M2,double>(
                              n
                              , k
                              , A_row_indices
                              , A_column_indices
                              , A_values
                              , x_values
                              , b_values
                              , max_iterations
                              , relative_tolerance
                              , absolute_tolerance
                              , preconditioner_choice
                              , solver_choice
                              ,residuals
                              );
  }
  
}//namespace util
